#include "hip/hip_runtime.h"
/*
    nvcc -arch=sm_30 bitonic_sort.cu
    ./a.out
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/*
  * Every thread gets exactly one value in the unsorted array.
  * must be powers of 2
  * (reduction) to help create better visualizations
*/
#define THREADS 4  // 2^2
#define BLOCKS 16  // 2^4
#define NUM_VALS THREADS*BLOCKS

FILE *fp;

void print_elapsed(clock_t start, clock_t stop) {
    double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float() {
    return (float) rand() / (float) RAND_MAX;
}

void array_print(float *arr, int length) {
    for (int i = 0; i < length; ++i) {
        printf("%1.3f ", arr[i]);
    }
    printf("\n");
}

/*
  * this function writes the current data in the arr
  * to the file pointer fp
  * (this function helps store the various configurations of the array during the sorting procedure)
*/
void array_write(float *arr, int length) {
    for (int i = 0; i < length; i++) {
        if (i + 1 < length) fprintf(fp, "%f, ", arr[i]);
        else fprintf(fp, "%f\n", arr[i]);
    }
}

/*
  * fill the array with randomized floating point values
*/
void array_fill(float *arr, int length) {
    srand(time(NULL));
    for (int i = 0; i < length; ++i) {
        arr[i] = random_float();
    }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k) {
    unsigned int i, ixj;  /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}


/**
 * Inplace bitonic sort using CUDA.
 */

void bitonic_sort(float *values) {
    float *print_values = (float *) malloc(NUM_VALS * sizeof(float));
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **) &dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);  /* Number of blocks   */
    dim3 threads(THREADS, 1);  /* Number of threads  */

    /* Major step */
    for (int k = 2; k <= NUM_VALS; k <<= 1) {
        /* Minor step */
        for (int j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step << < blocks, threads >> > (dev_values, j, k);
            hipMemcpy(print_values, dev_values, size, hipMemcpyDeviceToHost);
            array_write(print_values, NUM_VALS);
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

int main(void) {
    float *values = (float *) malloc(NUM_VALS * sizeof(float));
    array_fill(values, NUM_VALS);

    fp = fopen("tmp.csv", "w");
    if (fp == NULL) {
        puts("Couldn't open file");
        exit(0);
    }

    /*
      * add the header row to the csv file
    */
    for (int i = 0; i < NUM_VALS; i++) {
        if (i + 1 < NUM_VALS) fprintf(fp, "%d, ", i);
        else fprintf(fp, "%d\n", i);
    }

    bitonic_sort(values);

    fclose(fp);

    return 0;
}